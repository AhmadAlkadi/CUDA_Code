
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <stdlib.h>
#include <string>

__global__ void matrixMulKernel_1thread1elemen(float* A_d, float* B_d, float* C_d, int N, int M, int K){
    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;
    if(row < M && col < N){
        float sum = 0.0f;
        for(unsigned int i = 0; i <K; ++i){
            sum +=A_d[row*K + i] * B_d[i*K + col];
        }
        C_d[row*K + col] = sum;
    }
}

__global__ void matrixMulKernel_1thread1row(float* A_d, float* B_d, float* C_d, int N, int M, int K){
    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;


    float sum = 0.0f;
    if(row < M){
        for(unsigned int j = 0; j <N; ++j){
            for(unsigned int i = 0; i <N; ++i){
                sum +=A_d[row*N + i] * B_d[i*N + j];
                C_d[row*N + j] = sum;
            }
        }
    }
    
}


__global__ void matrixMulKernel_1thread1column(float* A_d, float* B_d, float* C_d, int N, int M, int K){
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

    float sum = 0.0f;
    if(col < N){
        for(unsigned int j = 0; j <N; ++j){
            for(unsigned int i = 0; i <N; ++i){
                sum +=A_d[j*N + i] * B_d[i*N + col];
                C_d[j*N + col] = sum;
            }
        }
    }
}

void basicSgemm_d_1thread1element (int m, int k, int n, const float *A_h, const float*B_h, float* C_h){
    float *A_d, *B_d, *C_d;
    auto start_time = std::chrono::high_resolution_clock::now();
    hipMalloc(&A_d,sizeof(float)*(m*k));
    hipMalloc(&B_d,sizeof(float)*(k*n));
    hipMalloc(&C_d,sizeof(float)*(m*n));
    auto end_time = std::chrono::high_resolution_clock::now();
    float time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;

    printf("    cudaMalloc: %24s%9.6fs \n", "",time_to_add_seconds);

    start_time = std::chrono::high_resolution_clock::now();
    hipMemcpy(A_d,A_h,sizeof(float)*(m*k),hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(B_d,B_h,sizeof(float)*(k*n),hipMemcpyKind::hipMemcpyHostToDevice);
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    
    printf("    cudaMemcpy: %24s%9.6fs \n", "", time_to_add_seconds);

    start_time = std::chrono::high_resolution_clock::now();
    int dimSize = 0;
    if(m > n){
        dimSize = std::ceil(m/1024.0f);
    }else{
        dimSize = std::ceil(n/1024.0f);
    }
    dim3 blockDim(dimSize,dimSize,1);
    dim3 gridDim(m,n,1);
    matrixMulKernel_1thread1elemen<<<gridDim,blockDim>>>(A_d,B_d,C_d,n,m,k);
    hipDeviceSynchronize();
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    printf("    matrixMulKernel_1thread1elemen<<<(%d,%d,%d),(%d,%d,%d)>>>: %9.6f \n",dimSize,dimSize,1,m,n,1,  time_to_add_seconds);
    hipDeviceSynchronize();

    start_time = std::chrono::high_resolution_clock::now();
    hipMemcpy(C_h,C_d,sizeof(float)*(m*n),hipMemcpyKind::hipMemcpyDeviceToHost);
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    printf("    CudaMemcpy: %24s%9.6fs \n", "", time_to_add_seconds);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

void basicSgemm_d_1thread1row  (int m, int k, int n, const float *A_h, const float*B_h, float* C_h){
    float *A_d, *B_d, *C_d;
    auto start_time = std::chrono::high_resolution_clock::now();
    hipMalloc(&A_d,sizeof(float)*(m*k));
    hipMalloc(&B_d,sizeof(float)*(k*n));
    hipMalloc(&C_d,sizeof(float)*(m*n));
    auto end_time = std::chrono::high_resolution_clock::now();
    float time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;

    printf("    cudaMalloc: %24s%9.6fs \n", "",time_to_add_seconds);

    start_time = std::chrono::high_resolution_clock::now();
    hipMemcpy(A_d,A_h,sizeof(float)*(m*k),hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(B_d,B_h,sizeof(float)*(k*n),hipMemcpyKind::hipMemcpyHostToDevice);
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    
    printf("    cudaMemcpy: %24s%9.6fs \n", "", time_to_add_seconds);

    start_time = std::chrono::high_resolution_clock::now();
    int dimSize = 0;
    if(m > n){
        dimSize = std::ceil(m/1024.0f);
    }else{
        dimSize = std::ceil(n/1024.0f);
    }
    dim3 blockDim(dimSize,dimSize,1);
    dim3 gridDim(m,n,1);
    matrixMulKernel_1thread1row<<<gridDim,blockDim>>>(A_d,B_d,C_d,n,m,k);
    hipDeviceSynchronize();
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    printf("    matrixMulKernel_1thread1row<<<(%d,%d,%d),(%d,%d,%d)>>>: %9.6f \n",dimSize,dimSize,1,m,n,1,  time_to_add_seconds);
    hipDeviceSynchronize();

    start_time = std::chrono::high_resolution_clock::now();
    hipMemcpy(C_h,C_d,sizeof(float)*(m*n),hipMemcpyKind::hipMemcpyDeviceToHost);
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    printf("    CudaMemcpy: %24s%9.6fs \n", "", time_to_add_seconds);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

void basicSgemm_d_1thread1column (int m, int k, int n, const float *A_h, const float*B_h, float* C_h){
    float *A_d, *B_d, *C_d;
    auto start_time = std::chrono::high_resolution_clock::now();
    hipMalloc(&A_d,sizeof(float)*(m*k));
    hipMalloc(&B_d,sizeof(float)*(k*n));
    hipMalloc(&C_d,sizeof(float)*(m*n));
    auto end_time = std::chrono::high_resolution_clock::now();
    float time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;

    printf("    cudaMalloc: %24s%9.6fs \n", "",time_to_add_seconds);

    start_time = std::chrono::high_resolution_clock::now();
    hipMemcpy(A_d,A_h,sizeof(float)*(m*k),hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(B_d,B_h,sizeof(float)*(k*n),hipMemcpyKind::hipMemcpyHostToDevice);
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    
    printf("    cudaMemcpy: %24s%9.6fs \n", "", time_to_add_seconds);

    start_time = std::chrono::high_resolution_clock::now();
    int dimSize = 0;
    if(m > n){
        dimSize = std::ceil(m/1024.0f);
    }else{
        dimSize = std::ceil(n/1024.0f);
    }
    dim3 blockDim(dimSize,dimSize,1);
    dim3 gridDim(m,n,1);
    matrixMulKernel_1thread1column<<<gridDim,blockDim>>>(A_d,B_d,C_d,n,m,k);
    hipDeviceSynchronize();
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    printf("    matrixMulKernel_1thread1column<<<(%d,%d,%d),(%d,%d,%d)>>>: %9.6f \n",dimSize,dimSize,1,m,n,1, time_to_add_seconds);
    hipDeviceSynchronize();

    start_time = std::chrono::high_resolution_clock::now();
    hipMemcpy(C_h,C_d,sizeof(float)*(m*n),hipMemcpyKind::hipMemcpyDeviceToHost);
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    printf("    CudaMemcpy: %24s%9.6fs \n", "", time_to_add_seconds);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main(int argc, char *argv[])
{
    hipDeviceSynchronize();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int m = std::stoi(argv[1]);
    int k = std::stoi(argv[2]);
    int n = std::stoi(argv[3]);
    float* A_h = (float*)malloc(sizeof(float)*(m*k));
    float* B_h = (float*)malloc(sizeof(float)*(k*n)); 
    float* C_h = (float*)malloc(sizeof(float)*(m*n));

    for(int i =0; i< m*k;i++){
        A_h[i] = rand()%100/100.0;
    }

    for(int i =0; i< k*n;i++){
        B_h[i] = rand()%100/100.0;
    }

    auto start_time = std::chrono::high_resolution_clock::now();
    for(int p = 0; p <m; p++){
        for(int i =0; i< m;i++){
            for(int j = 0; j< n; j++){
                C_h [i+(p*n)]+= A_h[j + (p*k)] * B_h[i + (j*k)];
            }
        }
    }

    auto end_time = std::chrono::high_resolution_clock::now();
    float time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;

    printf("VeccMult on CPU: %25s%9.6fs \n", "", time_to_add_seconds);

    start_time = std::chrono::high_resolution_clock::now();
    basicSgemm_d_1thread1element(m,k,n,A_h,B_h,C_h);
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    printf("matrixMulKernel_1thread1row on GPU: %25s%9.6fs \n", "", time_to_add_seconds);

    start_time = std::chrono::high_resolution_clock::now();
    basicSgemm_d_1thread1row(m,k,n,A_h,B_h,C_h);
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    printf("matrixMulKernel_1thread1row on GPU: %25s%9.6fs \n", "", time_to_add_seconds);

    start_time = std::chrono::high_resolution_clock::now();
    basicSgemm_d_1thread1column(m,k,n,A_h,B_h,C_h);
    end_time = std::chrono::high_resolution_clock::now();
    time_to_add_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6f;
    printf("matrixMulKernel_1thread1row on GPU: %25s%9.6fs \n", "", time_to_add_seconds);

    free(A_h);
    free(B_h);
    free(C_h);
  return 0;
}